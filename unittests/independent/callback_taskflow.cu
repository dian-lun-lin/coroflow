#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro/src/cuda/callback/taskflow/taro_callback_taskflow.hpp>
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>


//// --------------------------------------------------------
//// Testcase:: Independent
//// --------------------------------------------------------

void independent_cbtaskflow(size_t num_threads, size_t num_streams, size_t num_tasks) {
  taro::TaroCBTaskflow taro{num_threads, num_streams};

  std::vector<taro::TaskHandle> tasks(num_tasks);

  int* a;
  int* b; 
  int* c;
  size_t M{10};
  size_t K{10};
  size_t N{10};
  size_t BLOCK_SIZE = 32;
  dim3 dim_grid((N - 1) / BLOCK_SIZE + 1, (N - 1) / BLOCK_SIZE + 1, 1);
  dim3 dim_block(BLOCK_SIZE, BLOCK_SIZE, 1);

  hipMallocManaged(&a, M * K * sizeof(int));
  hipMallocManaged(&b, K * N * sizeof(int));
  hipMallocManaged(&c, M * N * num_tasks * sizeof(int));
  for(size_t i = 0; i < M * K; ++i) {
    a[i] = M + K;
  }
  for(size_t i = 0; i < K * N; ++i) {
    b[i] = K + N;
  }

  for(size_t i = 0; i < num_tasks; ++i) {
    tasks[i] = taro.emplace([&taro, i, a, b, c, M, K, N, dim_grid, dim_block]() -> taro::Coro {
      co_await taro.cuda_suspend([a, b, c, i, M, K, N, dim_grid, dim_block](hipStream_t st) {
        taro::cuda_matmul<<<dim_grid, dim_block, 0, st>>>(a, b, c + i * M * N, M, K, N);
      });

      for(size_t k = 0; k < M * N; ++k) {
        REQUIRE(c[k + i * M * N] == (int)(M + K) * (K + N) * K);
      }

      co_return;
    });
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  
  REQUIRE(hipFree(a) == hipSuccess);
  REQUIRE(hipFree(b) == hipSuccess);
  REQUIRE(hipFree(c) == hipSuccess);

}

TEST_CASE("independent.cbtaskflow.1thread.1stream.1task" * doctest::timeout(300)) {
  independent_cbtaskflow(1, 1, 1);
}

TEST_CASE("independent.cbtaskflow.2thread.1stream.3task" * doctest::timeout(300)) {
  independent_cbtaskflow(2, 1, 3);
}

TEST_CASE("independent.cbtaskflow.2thread.2stream.18task" * doctest::timeout(300)) {
  independent_cbtaskflow(2, 2, 18);
}

TEST_CASE("independent.cbtaskflow.2thread.3stream.18task" * doctest::timeout(300)) {
  independent_cbtaskflow(2, 3, 18);
}

TEST_CASE("independent.cbtaskflow.3thread.1stream.2task" * doctest::timeout(300)) {
  independent_cbtaskflow(3, 1, 2);
}

TEST_CASE("independent.cbtaskflow.3thread.2stream.4task" * doctest::timeout(300)) {
  independent_cbtaskflow(3, 2, 4);
}

TEST_CASE("independent.cbtaskflow.3thread.3stream.18task" * doctest::timeout(300)) {
  independent_cbtaskflow(3, 3, 18);
}

TEST_CASE("independent.cbtaskflow.4thread.1stream.1task" * doctest::timeout(300)) {
  independent_cbtaskflow(4, 1, 1);
}

TEST_CASE("independent.cbtaskflow.4thread.2stream.11task" * doctest::timeout(300)) {
  independent_cbtaskflow(4, 2, 11);
}

TEST_CASE("independent.cbtaskflow.4thread.8stream.38task" * doctest::timeout(300)) {
  independent_cbtaskflow(4, 8, 38);
}

TEST_CASE("independent.cbtaskflow.4thread.15stream.123task" * doctest::timeout(300)) {
  independent_cbtaskflow(4, 15, 123);
}
