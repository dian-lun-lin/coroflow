#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro/src/cuda/taro_v1.hpp>
#include <taro/src/cuda/taro_v2.hpp>
#include <taro/src/cuda/taro_v3.hpp>
#include <taro/src/cuda/taro_v4.hpp>
#include <taro/src/cuda/taro_v5.hpp>
#include <taro/src/cuda/taro_v6.hpp>
#include <taro/src/cuda/taro_v7.hpp>
#include <taro/src/cuda/taro_v8.hpp>
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}

void linear_chain_v1(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  hipStream_t st;
  hipStreamCreate(&st);

  taro::TaroV1 taro{num_threads};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      REQUIRE(*counter == t); 

      hipEvent_t finish;
      hipEventCreate(&finish);
      count<<<8, 32, 0, st>>>(counter);
      hipEventRecord(finish);

      auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      while(!isdone()) {
        co_await taro.suspend();
      }

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
  hipStreamDestroy(st);
}

TEST_CASE("linear_chain_v1.1thread" * doctest::timeout(300)) {
  linear_chain_v1(1, 1);
}

TEST_CASE("linear_chain_v1.2thread" * doctest::timeout(300)) {
  linear_chain_v1(99, 2);
}

TEST_CASE("linear_chain_v1.3thread" * doctest::timeout(300)) {
  linear_chain_v1(712, 3);
}

TEST_CASE("linear_chain_v1.4thread" * doctest::timeout(300)) {
  linear_chain_v1(443, 4);
}

TEST_CASE("linear_chain_v1.5thread" * doctest::timeout(300)) {
  linear_chain_v1(1111, 5);
}

TEST_CASE("linear_chain_v1.6thread" * doctest::timeout(300)) {
  linear_chain_v1(2, 6);
}

TEST_CASE("linear_chain_v1.7thread" * doctest::timeout(300)) {
  linear_chain_v1(5, 7);
}

TEST_CASE("linear_chain_v1.8thread" * doctest::timeout(300)) {
  linear_chain_v1(9211, 8);
}

void linear_chain_v2(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  hipStream_t st;
  hipStreamCreate(&st);

  taro::TaroV2 taro{num_threads};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      REQUIRE(*counter == t); 

      hipEvent_t finish;
      hipEventCreate(&finish);
      count<<<8, 32, 0, st>>>(counter);
      hipEventRecord(finish);

      auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      while(!isdone()) {
        co_await taro.suspend();
      }

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
  hipStreamDestroy(st);
}

TEST_CASE("linear_chain_v2.1thread" * doctest::timeout(300)) {
  linear_chain_v2(1, 1);
}

TEST_CASE("linear_chain_v2.2thread" * doctest::timeout(300)) {
  linear_chain_v2(99, 2);
}

TEST_CASE("linear_chain_v2.3thread" * doctest::timeout(300)) {
  linear_chain_v2(712, 3);
}

TEST_CASE("linear_chain_v2.4thread" * doctest::timeout(300)) {
  linear_chain_v2(443, 4);
}

TEST_CASE("linear_chain_v2.5thread" * doctest::timeout(300)) {
  linear_chain_v2(1111, 5);
}

TEST_CASE("linear_chain_v2.6thread" * doctest::timeout(300)) {
  linear_chain_v2(2, 6);
}

TEST_CASE("linear_chain_v2.7thread" * doctest::timeout(300)) {
  linear_chain_v2(5, 7);
}

TEST_CASE("linear_chain_v2.8thread" * doctest::timeout(300)) {
  linear_chain_v2(9211, 8);
}

void linear_chain_v3(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV3 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v3.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_v3(1, 1, 1);
}

TEST_CASE("linear_chain_v3.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_v3(99, 2, 2);
}

TEST_CASE("linear_chain_v3.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_v3(712, 3, 4);
}

TEST_CASE("linear_chain_v3.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_v3(443, 4, 8);
}

TEST_CASE("linear_chain_v3.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_v3(1111, 5, 2);
}

TEST_CASE("linear_chain_v3.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_v3(2, 6, 3);
}

TEST_CASE("linear_chain_v3.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_v3(5, 7, 1);
}

TEST_CASE("linear_chain_v3.8threads" * doctest::timeout(300)) {
  linear_chain_v3(9211, 8, 9);
}

void linear_chain_v4(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV4 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v4.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_v4(1, 1, 1);
}

TEST_CASE("linear_chain_v4.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_v4(99, 2, 2);
}

TEST_CASE("linear_chain_v4.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_v4(712, 3, 4);
}

TEST_CASE("linear_chain_v4.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_v4(443, 4, 8);
}

TEST_CASE("linear_chain_v4.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_v4(1111, 5, 2);
}

TEST_CASE("linear_chain_v4.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_v4(2, 6, 3);
}

TEST_CASE("linear_chain_v4.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_v4(5, 7, 1);
}

TEST_CASE("linear_chain_v4.8threads" * doctest::timeout(300)) {
  linear_chain_v4(9211, 8, 9);
}

void linear_chain_v5(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV5 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v5.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_v5(1, 1, 1);
}

TEST_CASE("linear_chain_v5.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_v5(99, 2, 2);
}

TEST_CASE("linear_chain_v5.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_v5(712, 3, 4);
}

TEST_CASE("linear_chain_v5.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_v5(443, 4, 8);
}

TEST_CASE("linear_chain_v5.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_v5(1111, 5, 2);
}

TEST_CASE("linear_chain_v5.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_v5(2, 6, 3);
}

TEST_CASE("linear_chain_v5.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_v5(5, 7, 1);
}

TEST_CASE("linear_chain_v5.8threads" * doctest::timeout(300)) {
  linear_chain_v5(9211, 8, 9);
}

void linear_chain_v6(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV6 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v6.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_v6(1, 1, 1);
}

TEST_CASE("linear_chain_v6.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_v6(99, 2, 2);
}

TEST_CASE("linear_chain_v6.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_v6(712, 3, 4);
}

TEST_CASE("linear_chain_v6.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_v6(443, 4, 8);
}

TEST_CASE("linear_chain_v6.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_v6(1111, 5, 2);
}

TEST_CASE("linear_chain_v6.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_v6(2, 6, 3);
}

TEST_CASE("linear_chain_v6.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_v6(5, 7, 1);
}

TEST_CASE("linear_chain_v6.8threads" * doctest::timeout(300)) {
  linear_chain_v6(9211, 8, 9);
}

void linear_chain_v7(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV7 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v7.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_v7(1, 1, 1);
}

TEST_CASE("linear_chain_v7.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_v7(99, 2, 2);
}

TEST_CASE("linear_chain_v7.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_v7(712, 3, 4);
}

TEST_CASE("linear_chain_v7.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_v7(443, 4, 8);
}

TEST_CASE("linear_chain_v7.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_v7(1111, 5, 2);
}

TEST_CASE("linear_chain_v7.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_v7(2, 6, 3);
}

TEST_CASE("linear_chain_v7.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_v7(5, 7, 1);
}

TEST_CASE("linear_chain_v7.8threads" * doctest::timeout(300)) {
  linear_chain_v7(9211, 8, 9);
}

void linear_chain_v8(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroV8 taro{num_threads};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_v8.1thread" * doctest::timeout(300)) {
  linear_chain_v8(1, 1);
}

TEST_CASE("linear_chain_v8.2thread" * doctest::timeout(300)) {
  linear_chain_v8(99, 2);
}

TEST_CASE("linear_chain_v8.3thread" * doctest::timeout(300)) {
  linear_chain_v8(712, 3);
}

TEST_CASE("linear_chain_v8.4thread" * doctest::timeout(300)) {
  linear_chain_v8(443, 4);
}

TEST_CASE("linear_chain_v8.5thread" * doctest::timeout(300)) {
  linear_chain_v8(1111, 5);
}

TEST_CASE("linear_chain_v8.6thread" * doctest::timeout(300)) {
  linear_chain_v8(2, 6);
}

TEST_CASE("linear_chain_v8.7thread" * doctest::timeout(300)) {
  linear_chain_v8(5, 7);
}

TEST_CASE("linear_chain_v8.8thread" * doctest::timeout(300)) {
  linear_chain_v8(9211, 8);
}
