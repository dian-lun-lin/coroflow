#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro/src/cuda/callback/taro_callback_v1.hpp>
#include <taro/src/cuda/callback/taro_callback_v2.hpp>
#include <taro/src/cuda/callback/taro_callback_v3.hpp>
#include <taro/src/cuda/poll/taro_poll_v1.hpp>
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}

void linear_chain_cbv1(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBV1 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_cbv1.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv1(1, 1, 1);
}

TEST_CASE("linear_chain_cbv1.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv1(99, 2, 2);
}

TEST_CASE("linear_chain_cbv1.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_cbv1(712, 3, 4);
}

TEST_CASE("linear_chain_cbv1.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_cbv1(443, 4, 8);
}

TEST_CASE("linear_chain_cbv1.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv1(1111, 5, 2);
}

TEST_CASE("linear_chain_cbv1.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_cbv1(2, 6, 3);
}

TEST_CASE("linear_chain_cbv1.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv1(5, 7, 1);
}

TEST_CASE("linear_chain_cbv1.8threads" * doctest::timeout(300)) {
  linear_chain_cbv1(9211, 8, 9);
}

void linear_chain_cbv2(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBV2 taro{num_threads};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_cbv2.1thread" * doctest::timeout(300)) {
  linear_chain_cbv2(1, 1);
}

TEST_CASE("linear_chain_cbv2.2thread" * doctest::timeout(300)) {
  linear_chain_cbv2(99, 2);
}

TEST_CASE("linear_chain_cbv2.3thread" * doctest::timeout(300)) {
  linear_chain_cbv2(712, 3);
}

TEST_CASE("linear_chain_cbv2.4thread" * doctest::timeout(300)) {
  linear_chain_cbv2(443, 4);
}

TEST_CASE("linear_chain_cbv2.5thread" * doctest::timeout(300)) {
  linear_chain_cbv2(1111, 5);
}

TEST_CASE("linear_chain_cbv2.6thread" * doctest::timeout(300)) {
  linear_chain_cbv2(2, 6);
}

TEST_CASE("linear_chain_cbv2.7thread" * doctest::timeout(300)) {
  linear_chain_cbv2(5, 7);
}

TEST_CASE("linear_chain_cbv2.8thread" * doctest::timeout(300)) {
  linear_chain_cbv2(9211, 8);
}


void linear_chain_cbv3(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBV3 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_cbv3.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv3(1, 1, 1);
}

TEST_CASE("linear_chain_cbv3.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv3(99, 2, 2);
}

TEST_CASE("linear_chain_cbv3.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_cbv3(712, 3, 4);
}

TEST_CASE("linear_chain_cbv3.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_cbv3(443, 4, 8);
}

TEST_CASE("linear_chain_cbv3.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv3(1111, 5, 2);
}

TEST_CASE("linear_chain_cbv3.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_cbv3(2, 6, 3);
}

TEST_CASE("linear_chain_cbv3.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv3(5, 7, 1);
}

TEST_CASE("linear_chain_cbv3.8threads" * doctest::timeout(300)) {
  linear_chain_cbv3(9211, 8, 9);
}

void linear_chain_pv1(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroPV1 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_pv1.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(1, 1, 1);
}

TEST_CASE("linear_chain_pv1.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(99, 2, 2);
}

TEST_CASE("linear_chain_pv1.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_pv1(712, 3, 4);
}

TEST_CASE("linear_chain_pv1.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_pv1(443, 4, 8);
}

TEST_CASE("linear_chain_pv1.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(1111, 5, 2);
}

TEST_CASE("linear_chain_pv1.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_pv1(2, 6, 3);
}

TEST_CASE("linear_chain_pv1.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(5, 7, 1);
}

TEST_CASE("linear_chain_pv1.8threads" * doctest::timeout(300)) {
  linear_chain_pv1(9211, 8, 9);
}
//void linear_chain_v1(size_t num_tasks, size_t num_threads) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //hipStream_t st;
  //hipStreamCreate(&st);

  //taro::TaroV1 taro{num_threads};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //hipEvent_t finish;
      //hipEventCreate(&finish);
      //count<<<8, 32, 0, st>>>(counter);
      //hipEventRecord(finish);

      //auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      //while(!isdone()) {
        //co_await taro.suspend();
      //}

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
  //hipStreamDestroy(st);
//}

//TEST_CASE("linear_chain_v1.1thread" * doctest::timeout(300)) {
  //linear_chain_v1(1, 1);
//}

//TEST_CASE("linear_chain_v1.2thread" * doctest::timeout(300)) {
  //linear_chain_v1(99, 2);
//}

//TEST_CASE("linear_chain_v1.3thread" * doctest::timeout(300)) {
  //linear_chain_v1(712, 3);
//}

//TEST_CASE("linear_chain_v1.4thread" * doctest::timeout(300)) {
  //linear_chain_v1(443, 4);
//}

//TEST_CASE("linear_chain_v1.5thread" * doctest::timeout(300)) {
  //linear_chain_v1(1111, 5);
//}

//TEST_CASE("linear_chain_v1.6thread" * doctest::timeout(300)) {
  //linear_chain_v1(2, 6);
//}

//TEST_CASE("linear_chain_v1.7thread" * doctest::timeout(300)) {
  //linear_chain_v1(5, 7);
//}

//TEST_CASE("linear_chain_v1.8thread" * doctest::timeout(300)) {
  //linear_chain_v1(9211, 8);
//}

//void linear_chain_v2(size_t num_tasks, size_t num_threads) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //hipStream_t st;
  //hipStreamCreate(&st);

  //taro::TaroV2 taro{num_threads};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //hipEvent_t finish;
      //hipEventCreate(&finish);
      //count<<<8, 32, 0, st>>>(counter);
      //hipEventRecord(finish);

      //auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      //while(!isdone()) {
        //co_await taro.suspend();
      //}

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
  //hipStreamDestroy(st);
//}

//TEST_CASE("linear_chain_v2.1thread" * doctest::timeout(300)) {
  //linear_chain_v2(1, 1);
//}

//TEST_CASE("linear_chain_v2.2thread" * doctest::timeout(300)) {
  //linear_chain_v2(99, 2);
//}

//TEST_CASE("linear_chain_v2.3thread" * doctest::timeout(300)) {
  //linear_chain_v2(712, 3);
//}

//TEST_CASE("linear_chain_v2.4thread" * doctest::timeout(300)) {
  //linear_chain_v2(443, 4);
//}

//TEST_CASE("linear_chain_v2.5thread" * doctest::timeout(300)) {
  //linear_chain_v2(1111, 5);
//}

//TEST_CASE("linear_chain_v2.6thread" * doctest::timeout(300)) {
  //linear_chain_v2(2, 6);
//}

//TEST_CASE("linear_chain_v2.7thread" * doctest::timeout(300)) {
  //linear_chain_v2(5, 7);
//}

//TEST_CASE("linear_chain_v2.8thread" * doctest::timeout(300)) {
  //linear_chain_v2(9211, 8);
//}

//void linear_chain_v3(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroV3 taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
//}

//TEST_CASE("linear_chain_v3.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v3(1, 1, 1);
//}

//TEST_CASE("linear_chain_v3.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v3(99, 2, 2);
//}

//TEST_CASE("linear_chain_v3.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_v3(712, 3, 4);
//}

//TEST_CASE("linear_chain_v3.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_v3(443, 4, 8);
//}

//TEST_CASE("linear_chain_v3.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v3(1111, 5, 2);
//}

//TEST_CASE("linear_chain_v3.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_v3(2, 6, 3);
//}

//TEST_CASE("linear_chain_v3.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v3(5, 7, 1);
//}

//TEST_CASE("linear_chain_v3.8threads" * doctest::timeout(300)) {
  //linear_chain_v3(9211, 8, 9);
//}

//void linear_chain_v4(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroV4 taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
//}

//TEST_CASE("linear_chain_v4.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v4(1, 1, 1);
//}

//TEST_CASE("linear_chain_v4.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v4(99, 2, 2);
//}

//TEST_CASE("linear_chain_v4.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_v4(712, 3, 4);
//}

//TEST_CASE("linear_chain_v4.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_v4(443, 4, 8);
//}

//TEST_CASE("linear_chain_v4.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v4(1111, 5, 2);
//}

//TEST_CASE("linear_chain_v4.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_v4(2, 6, 3);
//}

//TEST_CASE("linear_chain_v4.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v4(5, 7, 1);
//}

//TEST_CASE("linear_chain_v4.8threads" * doctest::timeout(300)) {
  //linear_chain_v4(9211, 8, 9);
//}
