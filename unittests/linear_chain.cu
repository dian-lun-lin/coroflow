#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro.hpp>
#include <taro/src/cuda/callback/taro_callback_v1.hpp>
#include <taro/src/cuda/callback/taro_callback_v2.hpp>
//#include <taro/src/cuda/callback/taro_callback_v3.hpp>
#include <taro/src/cuda/callback/taro_callback_taskflow.hpp>
//#include <taro/src/cuda/callback/taro_callback_taskflow_runtime.hpp>

#include "../benchmarks/boost_fiber/fiber.hpp"

#include <taro/src/cuda/poll/taro_poll_v1.hpp>
#include <taro/src/cuda/poll/taro_poll_v2.hpp>
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}

void linear_chain_cbv1(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBV1 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_cbv1.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv1(1, 1, 1);
}

TEST_CASE("linear_chain_cbv1.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv1(99, 2, 2);
}

TEST_CASE("linear_chain_cbv1.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_cbv1(712, 3, 4);
}

TEST_CASE("linear_chain_cbv1.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_cbv1(443, 4, 8);
}

TEST_CASE("linear_chain_cbv1.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbv1(1111, 5, 2);
}

TEST_CASE("linear_chain_cbv1.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_cbv1(2, 6, 3);
}

TEST_CASE("linear_chain_cbv1.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbv1(5, 7, 1);
}

TEST_CASE("linear_chain_cbv1.8threads" * doctest::timeout(300)) {
  linear_chain_cbv1(9211, 8, 9);
}

void linear_chain_cbv2(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBV2 taro{num_threads};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_cbv2.1thread" * doctest::timeout(300)) {
  linear_chain_cbv2(1, 1);
}

TEST_CASE("linear_chain_cbv2.2thread" * doctest::timeout(300)) {
  linear_chain_cbv2(99, 2);
}

TEST_CASE("linear_chain_cbv2.3thread" * doctest::timeout(300)) {
  linear_chain_cbv2(712, 3);
}

TEST_CASE("linear_chain_cbv2.4thread" * doctest::timeout(300)) {
  linear_chain_cbv2(443, 4);
}

TEST_CASE("linear_chain_cbv2.5thread" * doctest::timeout(300)) {
  linear_chain_cbv2(1111, 5);
}

TEST_CASE("linear_chain_cbv2.6thread" * doctest::timeout(300)) {
  linear_chain_cbv2(2, 6);
}

TEST_CASE("linear_chain_cbv2.7thread" * doctest::timeout(300)) {
  linear_chain_cbv2(5, 7);
}

TEST_CASE("linear_chain_cbv2.8thread" * doctest::timeout(300)) {
  linear_chain_cbv2(9211, 8);
}


//void linear_chain_cbv3(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroCBV3 taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
//}

//TEST_CASE("linear_chain_cbv3.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(1, 1, 1);
//}

//TEST_CASE("linear_chain_cbv3.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(99, 2, 2);
//}

//TEST_CASE("linear_chain_cbv3.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(712, 3, 4);
//}

//TEST_CASE("linear_chain_cbv3.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(443, 4, 8);
//}

//TEST_CASE("linear_chain_cbv3.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(1111, 5, 2);
//}

//TEST_CASE("linear_chain_cbv3.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(2, 6, 3);
//}

//TEST_CASE("linear_chain_cbv3.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_cbv3(5, 7, 1);
//}

//TEST_CASE("linear_chain_cbv3.8threads" * doctest::timeout(300)) {
  //linear_chain_cbv3(9211, 8, 9);
//}

void linear_chain_cbtaskflow(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroCBTaskflow taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
}

TEST_CASE("linear_chain_cbtaskflow.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(1, 1, 1);
}

TEST_CASE("linear_chain_cbtaskflow.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(99, 2, 2);
}

TEST_CASE("linear_chain_cbtaskflow.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(712, 3, 4);
}

TEST_CASE("linear_chain_cbtaskflow.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(443, 4, 8);
}

TEST_CASE("linear_chain_cbtaskflow.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(1111, 5, 2);
}

TEST_CASE("linear_chain_cbtaskflow.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(2, 6, 3);
}

TEST_CASE("linear_chain_cbtaskflow.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(5, 7, 1);
}

TEST_CASE("linear_chain_cbtaskflow.8threads" * doctest::timeout(300)) {
  linear_chain_cbtaskflow(9211, 8, 9);
}

//void linear_chain_cbtaskflowruntime(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroCBTaskflowRuntime taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(1, 1, 1);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(99, 2, 2);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(712, 3, 4);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(443, 4, 8);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(1111, 5, 2);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(2, 6, 3);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(5, 7, 1);
//}

//TEST_CASE("linear_chain_cbtaskflowruntime.8threads" * doctest::timeout(300)) {
  //linear_chain_cbtaskflowruntime(9211, 8, 9);
//}

void linear_chain_pv1(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroPV1 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_pv1.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(1, 1, 1);
}

TEST_CASE("linear_chain_pv1.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(99, 2, 2);
}

TEST_CASE("linear_chain_pv1.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_pv1(712, 3, 4);
}

TEST_CASE("linear_chain_pv1.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_pv1(443, 4, 8);
}

TEST_CASE("linear_chain_pv1.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(1111, 5, 2);
}

TEST_CASE("linear_chain_pv1.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_pv1(2, 6, 3);
}

TEST_CASE("linear_chain_pv1.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(5, 7, 1);
}

TEST_CASE("linear_chain_pv1.8threads" * doctest::timeout(300)) {
  linear_chain_pv1(9211, 8, 9);
}

void linear_chain_pv2(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroPV2 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_pv2.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv2(1, 1, 1);
}

TEST_CASE("linear_chain_pv2.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv2(99, 2, 2);
}

TEST_CASE("linear_chain_pv2.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_pv2(712, 3, 4);
}

TEST_CASE("linear_chain_pv2.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_pv2(443, 4, 8);
}

TEST_CASE("linear_chain_pv2.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv2(1111, 5, 2);
}

TEST_CASE("linear_chain_pv2.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_pv2(2, 6, 3);
}

TEST_CASE("linear_chain_pv2.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv2(5, 7, 1);
}

TEST_CASE("linear_chain_pv2.8threads" * doctest::timeout(300)) {
  linear_chain_pv2(9211, 8, 9);
}
//void linear_chain_v1(size_t num_tasks, size_t num_threads) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //hipStream_t st;
  //hipStreamCreate(&st);

  //taro::TaroV1 taro{num_threads};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //hipEvent_t finish;
      //hipEventCreate(&finish);
      //count<<<8, 32, 0, st>>>(counter);
      //hipEventRecord(finish);

      //auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      //while(!isdone()) {
        //co_await taro.suspend();
      //}

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
  //hipStreamDestroy(st);
//}

//TEST_CASE("linear_chain_v1.1thread" * doctest::timeout(300)) {
  //linear_chain_v1(1, 1);
//}

//TEST_CASE("linear_chain_v1.2thread" * doctest::timeout(300)) {
  //linear_chain_v1(99, 2);
//}

//TEST_CASE("linear_chain_v1.3thread" * doctest::timeout(300)) {
  //linear_chain_v1(712, 3);
//}

//TEST_CASE("linear_chain_v1.4thread" * doctest::timeout(300)) {
  //linear_chain_v1(443, 4);
//}

//TEST_CASE("linear_chain_v1.5thread" * doctest::timeout(300)) {
  //linear_chain_v1(1111, 5);
//}

//TEST_CASE("linear_chain_v1.6thread" * doctest::timeout(300)) {
  //linear_chain_v1(2, 6);
//}

//TEST_CASE("linear_chain_v1.7thread" * doctest::timeout(300)) {
  //linear_chain_v1(5, 7);
//}

//TEST_CASE("linear_chain_v1.8thread" * doctest::timeout(300)) {
  //linear_chain_v1(9211, 8);
//}

//void linear_chain_v2(size_t num_tasks, size_t num_threads) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //hipStream_t st;
  //hipStreamCreate(&st);

  //taro::TaroV2 taro{num_threads};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro, st]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //hipEvent_t finish;
      //hipEventCreate(&finish);
      //count<<<8, 32, 0, st>>>(counter);
      //hipEventRecord(finish);

      //auto isdone = [&finish]() { return hipEventQuery(finish) == hipSuccess;  };
      //while(!isdone()) {
        //co_await taro.suspend();
      //}

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
  //hipStreamDestroy(st);
//}

//TEST_CASE("linear_chain_v2.1thread" * doctest::timeout(300)) {
  //linear_chain_v2(1, 1);
//}

//TEST_CASE("linear_chain_v2.2thread" * doctest::timeout(300)) {
  //linear_chain_v2(99, 2);
//}

//TEST_CASE("linear_chain_v2.3thread" * doctest::timeout(300)) {
  //linear_chain_v2(712, 3);
//}

//TEST_CASE("linear_chain_v2.4thread" * doctest::timeout(300)) {
  //linear_chain_v2(443, 4);
//}

//TEST_CASE("linear_chain_v2.5thread" * doctest::timeout(300)) {
  //linear_chain_v2(1111, 5);
//}

//TEST_CASE("linear_chain_v2.6thread" * doctest::timeout(300)) {
  //linear_chain_v2(2, 6);
//}

//TEST_CASE("linear_chain_v2.7thread" * doctest::timeout(300)) {
  //linear_chain_v2(5, 7);
//}

//TEST_CASE("linear_chain_v2.8thread" * doctest::timeout(300)) {
  //linear_chain_v2(9211, 8);
//}

//void linear_chain_v3(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroV3 taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
//}

//TEST_CASE("linear_chain_v3.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v3(1, 1, 1);
//}

//TEST_CASE("linear_chain_v3.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v3(99, 2, 2);
//}

//TEST_CASE("linear_chain_v3.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_v3(712, 3, 4);
//}

//TEST_CASE("linear_chain_v3.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_v3(443, 4, 8);
//}

//TEST_CASE("linear_chain_v3.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_v3(1111, 5, 2);
//}

//TEST_CASE("linear_chain_v3.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_v3(2, 6, 3);
//}

//TEST_CASE("linear_chain_v3.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_v3(5, 7, 1);
//}

//TEST_CASE("linear_chain_v3.8threads" * doctest::timeout(300)) {
  //linear_chain_v3(9211, 8, 9);
//}

//void linear_chain_taskflow(size_t num_tasks, size_t num_threads, size_t num_streams) {
  //int* counter;
  //hipMallocManaged(&counter, sizeof(int));

  //taro::TaroTaskflow taro{num_threads, num_streams};
  //std::vector<taro::TaskHandle> _tasks(num_tasks);

  //for(size_t t = 0; t < num_tasks; ++t) {
    //_tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      //REQUIRE(*counter == t); 

      //co_await taro.cuda_suspend([counter](hipStream_t st) {
        //count<<<8, 32, 0, st>>>(counter);
      //});

      //REQUIRE(*counter == t + 1); 
    //});
  //}

  //for(size_t t = 0; t < num_tasks - 1; ++t) {
    //_tasks[t].precede(_tasks[t + 1]);
  //}

  //REQUIRE(taro.is_DAG());
  //taro.schedule();
  //taro.wait(); 
//}

//TEST_CASE("linear_chain_taskflow.1thread.1stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(1, 1, 1);
//}

//TEST_CASE("linear_chain_taskflow.2thread.2stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(99, 2, 2);
//}

//TEST_CASE("linear_chain_taskflow.3thread.4stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(712, 3, 4);
//}

//TEST_CASE("linear_chain_taskflow.4thread.8stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(443, 4, 8);
//}

//TEST_CASE("linear_chain_taskflow.5thread.2stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(1111, 5, 2);
//}

//TEST_CASE("linear_chain_taskflow.6thread.3stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(2, 6, 3);
//}

//TEST_CASE("linear_chain_taskflow.7thread.1stream" * doctest::timeout(300)) {
  //linear_chain_taskflow(5, 7, 1);
//}

//TEST_CASE("linear_chain_taskflow.8threads" * doctest::timeout(300)) {
  //linear_chain_taskflow(9211, 8, 9);
//}

void linear_chain_fiber(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  FiberTaskScheduler ft_sched{num_threads};
  std::vector<FiberTaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = ft_sched.emplace([t, counter, &ft_sched]() {
      REQUIRE(*counter == t); 
      hipStream_t st;
      hipStreamCreateWithFlags(&st, hipStreamNonBlocking);

      count<<<8, 32, 0, st>>>(counter);

      boost::fibers::cuda::waitfor_all(st);
      hipStreamDestroy(st);


      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  ft_sched.schedule();
  ft_sched.wait(); 
}

TEST_CASE("linear_chain_fiber.2thread" * doctest::timeout(300)) {
  linear_chain_fiber(99, 2);
}

TEST_CASE("linear_chain_fiber.3thread" * doctest::timeout(300)) {
  linear_chain_fiber(712, 3);
}

TEST_CASE("linear_chain_fiber.4thread" * doctest::timeout(300)) {
  linear_chain_fiber(443, 4);
}

TEST_CASE("linear_chain_fiber.5thread" * doctest::timeout(300)) {
  linear_chain_fiber(1111, 5);
}

TEST_CASE("linear_chain_fiber.6thread" * doctest::timeout(300)) {
  linear_chain_fiber(2, 6);
}

TEST_CASE("linear_chain_fiber.7thread" * doctest::timeout(300)) {
  linear_chain_fiber(5, 7);
}

TEST_CASE("linear_chain_fiber.8thread" * doctest::timeout(300)) {
  linear_chain_fiber(9211, 8);
}

