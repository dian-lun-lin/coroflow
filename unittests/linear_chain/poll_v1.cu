#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro.hpp>
#include <taro/src/cuda/poll/v1/taro_poll_v1.hpp>
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}
void linear_chain_pv1(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroPV1 taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_pv1.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(1, 1, 1);
}

TEST_CASE("linear_chain_pv1.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(99, 2, 2);
}

TEST_CASE("linear_chain_pv1.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_pv1(712, 3, 4);
}

TEST_CASE("linear_chain_pv1.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_pv1(443, 4, 8);
}

TEST_CASE("linear_chain_pv1.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_pv1(1111, 5, 2);
}

TEST_CASE("linear_chain_pv1.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_pv1(2, 6, 3);
}

TEST_CASE("linear_chain_pv1.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_pv1(5, 7, 1);
}

TEST_CASE("linear_chain_pv1.8threads" * doctest::timeout(300)) {
  linear_chain_pv1(9211, 8, 9);
}
