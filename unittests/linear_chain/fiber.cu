#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include <taro.hpp>
#include "../benchmarks/boost_fiber/fiber.hpp"
#include <taro/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}

void linear_chain_fiber(size_t num_tasks, size_t num_threads) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  FiberTaskScheduler ft_sched{num_threads};
  std::vector<FiberTaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = ft_sched.emplace([t, counter, &ft_sched]() {
      REQUIRE(*counter == t); 
      hipStream_t st;
      hipStreamCreateWithFlags(&st, hipStreamNonBlocking);

      count<<<8, 32, 0, st>>>(counter);

      boost::fibers::cuda::waitfor_all(st);
      hipStreamDestroy(st);


      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  ft_sched.schedule();
  ft_sched.wait(); 
}

TEST_CASE("linear_chain_fiber.2thread" * doctest::timeout(300)) {
  linear_chain_fiber(99, 2);
}

TEST_CASE("linear_chain_fiber.3thread" * doctest::timeout(300)) {
  linear_chain_fiber(712, 3);
}

TEST_CASE("linear_chain_fiber.4thread" * doctest::timeout(300)) {
  linear_chain_fiber(443, 4);
}

TEST_CASE("linear_chain_fiber.5thread" * doctest::timeout(300)) {
  linear_chain_fiber(1111, 5);
}

TEST_CASE("linear_chain_fiber.6thread" * doctest::timeout(300)) {
  linear_chain_fiber(2, 6);
}

TEST_CASE("linear_chain_fiber.7thread" * doctest::timeout(300)) {
  linear_chain_fiber(5, 7);
}

TEST_CASE("linear_chain_fiber.8thread" * doctest::timeout(300)) {
  linear_chain_fiber(9211, 8);
}

