#include "hip/hip_runtime.h"
#define DOCTEST_CONFIG_IMPLEMENT_WITH_MAIN
#include <doctest.h>
#include "../../taro_taskflow/taro_taskflow.hpp"
#include <taro/algorithm/cuda.hpp>
#include <vector>
#include <algorithm>
#include <numeric>

// --------------------------------------------------------
// Testcase:: Linear chain
// --------------------------------------------------------

// o - o - o - o

template <typename T>
__global__
void count(T* count) {
  ++(*count);
}

void linear_chain_taskflow(size_t num_tasks, size_t num_threads, size_t num_streams) {
  int* counter;
  hipMallocManaged(&counter, sizeof(int));

  taro::TaroTaskflow taro{num_threads, num_streams};
  std::vector<taro::TaskHandle> _tasks(num_tasks);

  for(size_t t = 0; t < num_tasks; ++t) {
    _tasks[t] = taro.emplace([t, counter, &taro]() -> taro::Coro {
      REQUIRE(*counter == t); 

      co_await taro.cuda_suspend([counter](hipStream_t st) {
        count<<<8, 32, 0, st>>>(counter);
      });

      REQUIRE(*counter == t + 1); 
    });
  }

  for(size_t t = 0; t < num_tasks - 1; ++t) {
    _tasks[t].precede(_tasks[t + 1]);
  }

  REQUIRE(taro.is_DAG());
  taro.schedule();
  taro.wait(); 
}

TEST_CASE("linear_chain_taskflow.1thread.1stream" * doctest::timeout(300)) {
  linear_chain_taskflow(1, 1, 1);
}

TEST_CASE("linear_chain_taskflow.2thread.2stream" * doctest::timeout(300)) {
  linear_chain_taskflow(99, 2, 2);
}

TEST_CASE("linear_chain_taskflow.3thread.4stream" * doctest::timeout(300)) {
  linear_chain_taskflow(712, 3, 4);
}

TEST_CASE("linear_chain_taskflow.4thread.8stream" * doctest::timeout(300)) {
  linear_chain_taskflow(443, 4, 8);
}

TEST_CASE("linear_chain_taskflow.5thread.2stream" * doctest::timeout(300)) {
  linear_chain_taskflow(1111, 5, 2);
}

TEST_CASE("linear_chain_taskflow.6thread.3stream" * doctest::timeout(300)) {
  linear_chain_taskflow(2, 6, 3);
}

TEST_CASE("linear_chain_taskflow.7thread.1stream" * doctest::timeout(300)) {
  linear_chain_taskflow(5, 7, 1);
}

TEST_CASE("linear_chain_taskflow.8threads" * doctest::timeout(300)) {
  linear_chain_taskflow(9211, 8, 9);
}

