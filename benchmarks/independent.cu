#include "hip/hip_runtime.h"
#include <coroflow/src/cuda/coroflow_v1.hpp>
#include <coroflow/src/cuda/coroflow_v2.hpp>
#include <coroflow/src/cuda/coroflow_v3.hpp>
#include <coroflow/src/cuda/coroflow_v4.hpp>
#include <coroflow/src/cuda/coroflow_v5.hpp>
#include <coroflow/src/cuda/coroflow_v6.hpp>
#include <coroflow/src/cuda/algorithm.hpp>
#include <vector>
#include <algorithm>
#include <numeric>
#include <cassert>
#include <cstring>
#include <chrono>


// GPU kernel
__global__ void cuda_sleep(
   int ms
) {
  for (int i = 0; i < ms; i++) {
    __nanosleep(1000000U);
  }
}


// CPU task
void cpu_sleep(
  int ms
) {
  std::this_thread::sleep_for(std::chrono::milliseconds(ms));
}

// =============================================================
// 
// benchmark: independent
//
// =============================================================

// without coroutine
// one task, one stream
void func(
  size_t num_threads, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {

  std::chrono::time_point<std::chrono::steady_clock> task_tic;
  std::chrono::time_point<std::chrono::steady_clock> task_toc;

  cf::CoroflowV1 cf{num_threads};
  std::vector<cf::TaskHandle> tasks(num_tasks);
  std::vector<hipEvent_t>  events(num_tasks);
  std::vector<hipStream_t> streams(num_tasks);
  for(size_t i = 0; i < num_tasks; ++i) {
    hipStreamCreate(&streams[i]);
    hipEventCreate(&events[i]);
  }

  // emplace tasks
  for(size_t t = 0; t < num_tasks; ++t) {
    tasks[t] = cf.emplace(
      [&streams, &events, t, chain_size, cpu_ms, gpu_ms](){
      for(size_t i = 0; i < chain_size; i++) {
        // cpu task
        cpu_sleep(cpu_ms);

        // gpu task
        cuda_sleep<<<8, 32, 0, streams[t]>>>(gpu_ms);
        hipEventRecord(events[t]);
        hipEventSynchronize(events[t]);
      }
    });
  }

  assert(cf.is_DAG());

  task_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  task_toc = std::chrono::steady_clock::now();
  auto task_dur = std::chrono::duration_cast<std::chrono::milliseconds>(task_toc - task_tic).count();
  std::cout << "function time: " << task_dur << "ms\n";

  for(auto& st: streams) {
    hipStreamDestroy(st);
  }
}

// without callback
// one task, one stream
void coro_v1(
  size_t num_threads, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV1 cf{num_threads};
  std::vector<cf::TaskHandle> tasks(num_tasks);
  std::vector<hipStream_t> streams(num_tasks);
  std::vector<hipEvent_t> events(num_tasks);
  for(size_t i = 0; i < num_tasks; ++i) {
    hipStreamCreate(&streams[i]);
    hipEventCreate(&events[i]);
  }

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, &streams, &events, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          cuda_sleep<<<8, 32, 0, streams[c]>>>(gpu_ms);
          hipEventRecord(events[i]);
          auto isdone = [&events, i]() { return hipEventQuery(events[i]) == hipSuccess;  };
          while(!isdone()) {
            co_await cf.suspend();
          }
          
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v1 time: " << coro_dur << "ms\n";
}

// streams are handled by users
// one task, one stream
void coro_v2(
  size_t num_threads, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV2 cf{num_threads};
  std::vector<cf::TaskHandle> tasks(num_tasks);
  std::vector<hipStream_t> streams(num_tasks);
  for(size_t i = 0; i < num_tasks; ++i) {
    hipStreamCreate(&streams[i]);
  }

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, &streams, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          cuda_sleep<<<8, 32, 0, streams[c]>>>(gpu_ms);
          co_await cf.cuda_suspend(streams[c]);
          
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v2 time: " << coro_dur << "ms\n";
}


// M CPU threads, N GPU streams
void coro_v3(
  size_t num_threads, 
  size_t num_streams, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV3 cf{num_threads, num_streams};
  std::vector<cf::TaskHandle> tasks(num_tasks);

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          co_await cf.cuda_suspend([gpu_ms](hipStream_t st) {
            cuda_sleep<<<8, 32, 0, st>>>(gpu_ms);
          });
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v3 time: " << coro_dur << "ms\n";
}

// M CPU threads, N GPU streams
// work-stealing approach
void coro_v4(
  size_t num_threads, 
  size_t num_streams, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV4 cf{num_threads, num_streams};
  std::vector<cf::TaskHandle> tasks(num_tasks);

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          co_await cf.cuda_suspend([gpu_ms](hipStream_t st) {
            cuda_sleep<<<8, 32, 0, st>>>(gpu_ms);
          });
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v4 time: " << coro_dur << "ms\n";
}

// M CPU threads, N GPU streams
// work-stealing approach
void coro_v5(
  size_t num_threads, 
  size_t num_streams, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV5 cf{num_threads, num_streams};
  std::vector<cf::TaskHandle> tasks(num_tasks);

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          co_await cf.cuda_suspend([gpu_ms](hipStream_t st) {
            cuda_sleep<<<8, 32, 0, st>>>(gpu_ms);
          });
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v5 time: " << coro_dur << "ms\n";
}

// M CPU threads, N GPU streams
// work-stealing approach
void coro_v6(
  size_t num_threads, 
  size_t num_streams, 
  size_t num_tasks, 
  size_t chain_size, 
  int cpu_ms, 
  int gpu_ms
) {
  std::chrono::time_point<std::chrono::steady_clock> coro_tic;
  std::chrono::time_point<std::chrono::steady_clock> coro_toc;

  cf::CoroflowV5 cf{num_threads, num_streams};
  std::vector<cf::TaskHandle> tasks(num_tasks);

  // emplace tasks
  for(size_t c = 0; c < num_tasks; ++c) {
      tasks[c] = cf.emplace([&cf, c, chain_size, cpu_ms, gpu_ms]() -> cf::Coro {
        for(size_t i = 0; i < chain_size; i++) {
          // cpu task
          cpu_sleep(cpu_ms);

          // gpu task
          co_await cf.cuda_suspend([gpu_ms](hipStream_t st) {
            cuda_sleep<<<8, 32, 0, st>>>(gpu_ms);
          });
        }
        co_return;
      });
  }

  assert(cf.is_DAG());

  coro_tic = std::chrono::steady_clock::now();
  cf.schedule();
  cf.wait();
  coro_toc = std::chrono::steady_clock::now();
  auto coro_dur = std::chrono::duration_cast<std::chrono::milliseconds>(coro_toc - coro_tic).count();
  std::cout << "coroflow v6 time: " << coro_dur << "ms\n";
}


int main(int argc, char* argv[]) {
  if(argc != 8) {
    std::cerr << "usage: ./bin/independent mode num_threads num_streams num_tasks chain_size cpu_ms gpu_ms\n";
    std::cerr << "mode should be 0, 1, 2, 3, 4, 5, 6, or 7\n";
    std::cerr << "0: function, 1: coroflow v1, 2: corflow v2, 3: corflow v3... 7: all \n";
    std::exit(EXIT_FAILURE);
  }
  size_t mode = std::atoi(argv[1]);
  size_t num_threads = std::atoi(argv[2]);
  size_t num_streams = std::atoi(argv[3]);
  size_t num_tasks = std::atoi(argv[4]);
  size_t chain_size = std::atoi(argv[5]);
  int cpu_ms = std::atoi(argv[6]);
  int gpu_ms = std::atoi(argv[7]);
  std::cout << "(mode, num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms): (" 
            << mode        << ", "
            << num_threads << ", "
            << num_streams << ", "
            << num_tasks   << ", "
            << chain_size  << ", "
            << cpu_ms      << ", "
            << gpu_ms      << "):\n";

  if(mode == 0) {
    std::cout << "function...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    func(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 1) {
    std::cout << "coroflow v1...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    coro_v1(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 2) {
    std::cout << "coroflow v2...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    coro_v2(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 3) {
    std::cout << "coroflow v3...\n";
    coro_v3(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 4) {
    std::cout << "coroflow v4...\n";
    coro_v4(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 5) {
    std::cout << "coroflow v5...\n";
    coro_v5(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 6) {
    std::cout << "coroflow v6...\n";
    coro_v6(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else if(mode == 7) {
    std::cout << "all...\n\n";
    std::cout << "function...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    func(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
    std::cout << "\n";

    // without callback
    std::cout << "coroflow v1...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    coro_v1(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
    std::cout << "\n";

    // streams are handled by users
    std::cout << "coroflow v2...\n";
    std::cout << "igonre num_streams... each task has its own stream\n";
    coro_v2(num_threads, num_tasks, chain_size, cpu_ms, gpu_ms);
    std::cout << "\n";

    std::cout << "coroflow v3...\n";
    coro_v3(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);

    std::cout << "coroflow v4...\n";
    coro_v4(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);

    std::cout << "coroflow v5...\n";
    coro_v5(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);

    std::cout << "coroflow v6...\n";
    coro_v6(num_threads, num_streams, num_tasks, chain_size, cpu_ms, gpu_ms);
  }
  else {
    std::cerr << "mode should be 0, 1, 2, 3, 4, 5, 6, or 7\n";
    std::cerr << "0: function, 1: coroflow v1, 2: corflow v2, 3: corflow v3... 7: all \n";
    std::exit(EXIT_FAILURE);
  }


  std::cout << "\n";
}


